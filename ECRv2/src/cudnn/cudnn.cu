#include "hip/hip_fp16.h"

#include "cudnn_util.h"

using namespace std;


bool runCUDNN(Matrix &input, Matrix &kernel, HostData &host, int stride_width,
            int batch_size) {

    if (!host.input.data || host.kernel.data) {
        std::cerr << "Input or kernel is not allocated on the host\n";
        return false;
    }

    hipdnnHandle_t cudnn;
    Matrix output(host.output.feature_width,host.output.feature_height);
    CudaTimer timer;

    timer.startTiming();

    // Allocating memory for input and kernel on GPU
    checkCudaErrors(
      hipMalloc(&input.data, input.feature_width * input.feature_height * sizeof(float)));
    checkCudaErrors(
      hipMalloc(&kernel.data, kernel.feature_width * kernel.feature_height * sizeof(float)));
    checkCudaErrors(
      hipMalloc(&output.data, output.feature_width * output.feature_height * sizeof(float)));

    const int in_n = batch_size;
    const int in_c = 1;
    // in_h => feature_height
    // in_w => feature_width
    const int in_size = input.feature_height * input.feature_width * in_c * in_n;

    const int filt_k = 1;
    const int filt_c = 1;
    // filt_h => kernel height
    // filt_width => kernel_width
    const int file_size = kernel.feature_height * kernel.feature_width * filt_c * filt_k;
    
    CudaTimer timer;
    timer.startTiming();

    hipdnnTensorDescriptor_t in_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
        in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
        in_n, in_c, input.feature_height, input.feature_width));

    float *in_data;
    CUDA_CALL(hipMalloc(
        &in_data, in_n * in_c * input.feature_height * input.feature_width * sizeof(float)));

    hipdnnFilterDescriptor_t filt_desc;
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(
        filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW,
        filt_k, filt_c, kernel.feature_height, kernel.feature_width));

    float *filt_data;
    CUDA_CALL(hipMalloc(
        &filt_data, filt_k * filt_c * kernel.feature_height * kernel.feature_width * sizeof(float)));

    // convolution
    const int pad_h = 0;
    const int pad_w = 0;
    const int str_h = 1;
    const int str_w = 1;
    const int dil_h = 1;
    const int dil_w = 1;

    hipdnnConvolutionDescriptor_t conv_desc;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
        conv_desc,
        pad_h, pad_w, str_h, str_w, dil_h, dil_w,
        HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT));

    // hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH);
    // output
    int out_n;
    int out_c;
    int out_h;
    int out_w;

    CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(
        conv_desc, in_desc, filt_desc,
        &out_n, &out_c, &out_h, &out_w));

    hipdnnTensorDescriptor_t out_desc;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(
        out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT,
        out_n, out_c, out_h, out_w));

    float *out_data;
    CUDA_CALL(hipMalloc(
        &out_data, out_n * out_c * out_h * out_w * sizeof(float)));

    // algorithm
    hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;

    // = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
    // = HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED;
    // = HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD;
    // = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
    // = HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING;
    // = HIPDNN_CONVOLUTION_FWD_ALGO_GEMM;

    // CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
    //     cudnn,
    //     in_desc, filt_desc, conv_desc, out_desc,
    //     HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

    // workspace
    size_t ws_size;
    CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
        cudnn, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));

    float *ws_data;
    CUDA_CALL(hipMalloc(&ws_data, ws_size));

    // perform
    float alpha = 1.f;
    float beta = 0.f;

    hipMemcpy(in_data, input.data, in_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(filt_data, kernel.data, file_size * sizeof(float), hipMemcpyHostToDevice);

    CUDNN_CALL(hipdnnConvolutionForward(
        cudnn,
        &alpha, in_desc, in_data, filt_desc, filt_data,
        conv_desc, algo, ws_data, ws_size,
        &beta, out_desc, out_data));


    int result_size = out_n * out_c * out_h * out_w;
    float *result = new float[result_size];
    hipMemcpy(result, out_data, result_size * sizeof(float), hipMemcpyDeviceToHost);

    // finalizing
    CUDA_CALL(hipFree(ws_data));
    CUDA_CALL(hipFree(out_data));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(out_desc));
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CUDA_CALL(hipFree(filt_data));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(filt_desc));
    CUDA_CALL(hipFree(in_data));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(in_desc));
    CUDNN_CALL(hipdnnDestroy(cudnn));

    timer.stopTiming();
    host.time = timer.getElapsedTime();

    return true;
}