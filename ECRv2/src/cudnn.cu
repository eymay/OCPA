#include "hip/hip_fp16.h"
#include "hipDNN.h"

#include "ocpa.h"
#include "ocpa_cuda.h"

using namespace std;

bool runCUDNN(Matrix &input, Matrix &kernel, HostData &host, int stride_width,
              int batch_size) {

  if (!host.input.data || host.kernel.data) {
    std::cerr << "Input or kernel is not allocated on the host\n";
    return false;
  }

  hipdnnHandle_t cudnn;
  CUDNN_CALL(hipdnnCreate(&cudnn));

  Matrix output(host.output.width, host.output.height);
  CudaTimer timer;

  timer.startTiming();

  // Allocating memory for input and kernel on GPU
  checkCudaErrors(
      hipMalloc(&input.data, input.width * input.height * sizeof(float)));
  checkCudaErrors(
      hipMalloc(&kernel.data, kernel.width * kernel.height * sizeof(float)));
  checkCudaErrors(
      hipMalloc(&output.data, output.width * output.height * sizeof(float)));

  const int in_n = batch_size;
  constexpr int in_c = 1;
  // in_h => height
  // in_w => width
  const int in_size = input.height * input.width * in_c * in_n;

  constexpr int filt_k = 1;
  constexpr int filt_c = 1;
  // filt_h => kernel height
  // filt_width => kernel_width
  const int file_size = kernel.height * kernel.width * filt_c * filt_k;

  hipdnnTensorDescriptor_t in_desc;
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&in_desc));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(in_desc, HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT, in_n, in_c,
                                        input.height, input.width));

  float *in_data;
  CUDA_CALL(hipMalloc(&input.data, in_n * in_c * input.height * input.width *
                                     sizeof(float)));

  hipdnnFilterDescriptor_t filt_desc;
  CUDNN_CALL(hipdnnCreateFilterDescriptor(&filt_desc));
  CUDNN_CALL(hipdnnSetFilter4dDescriptor(filt_desc, HIPDNN_DATA_FLOAT,
                                        HIPDNN_TENSOR_NCHW, filt_k, filt_c,
                                        kernel.height, kernel.width));

  float *filt_data;
  CUDA_CALL(hipMalloc(&kernel.data, filt_k * filt_c * kernel.height *
                                       kernel.width * sizeof(float)));

  // convolution
  const int pad_h = 0;
  const int pad_w = 0;
  const int str_h = 1;
  const int str_w = 1;
  const int dil_h = 1;
  const int dil_w = 1;

  hipdnnConvolutionDescriptor_t conv_desc;
  CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&conv_desc));
  CUDNN_CALL(hipdnnSetConvolution2dDescriptor(
      conv_desc, pad_h, pad_w, str_h, str_w, dil_h, dil_w, HIPDNN_CONVOLUTION,
      HIPDNN_DATA_FLOAT));

  // hipdnnSetConvolutionMathType(conv_desc, HIPDNN_TENSOR_OP_MATH);
  // output
  int out_n;
  int out_c;
  int out_h;
  int out_w;

  CUDNN_CALL(hipdnnGetConvolution2dForwardOutputDim(
      conv_desc, in_desc, filt_desc, &out_n, &out_c, &out_h, &out_w));

  hipdnnTensorDescriptor_t out_desc;
  CUDNN_CALL(hipdnnCreateTensorDescriptor(&out_desc));
  CUDNN_CALL(hipdnnSetTensor4dDescriptor(out_desc, HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT, out_n, out_c, out_h,
                                        out_w));

  float *out_data;
  CUDA_CALL(
      hipMalloc(&output.data, out_n * out_c * out_h * out_w * sizeof(float)));

  // algorithm
  hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;

  // = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;
  // = HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED;
  // = HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD;
  // = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
  // = HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING;
  // = HIPDNN_CONVOLUTION_FWD_ALGO_GEMM;

  // CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(
  //     cudnn,
  //     in_desc, filt_desc, conv_desc, out_desc,
  //     HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo));

  // workspace
  size_t ws_size;
  CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(
      cudnn, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size));

  float *ws_data;
  CUDA_CALL(hipMalloc(&ws_data, ws_size));

  // perform
  float alpha = 1.f;
  float beta = 0.f;

  hipMemcpy(in_data, input.data, in_size * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(filt_data, kernel.data, file_size * sizeof(float),
             hipMemcpyHostToDevice);

  CUDNN_CALL(hipdnnConvolutionForward(cudnn, &alpha, in_desc, in_data, filt_desc,
                                     filt_data, conv_desc, algo, ws_data,
                                     ws_size, &beta, out_desc, out_data));

  int result_size = out_n * out_c * out_h * out_w;
  float *result = new float[result_size];
  hipMemcpy(result, out_data, result_size * sizeof(float),
             hipMemcpyDeviceToHost);

  // finalizing
  CUDA_CALL(hipFree(ws_data));
  CUDA_CALL(hipFree(out_data));
  CUDNN_CALL(hipdnnDestroyTensorDescriptor(out_desc));
  CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(conv_desc));
  CUDA_CALL(hipFree(filt_data));
  CUDNN_CALL(hipdnnDestroyFilterDescriptor(filt_desc));
  CUDA_CALL(hipFree(in_data));
  CUDNN_CALL(hipdnnDestroyTensorDescriptor(in_desc));
  CUDNN_CALL(hipdnnDestroy(cudnn));

  timer.stopTiming();
  host.time = timer.getElapsedTime();

  return true;
}
